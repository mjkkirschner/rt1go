#include "hip/hip_runtime.h"
#include "render.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__
void renderPixel(int n,int *rout, int *gout, int *bout){
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride)
    {
        rout[i] = index;
        gout[i] = index;
        bout[i] = index;
    }
}



void wrapper(int width, int height, int *xs, int *ys, int *rout, int *gout, int *bout){
    
    int N = width * height;
    // Allocate Unified Memory – accessible from CPU or GPU
int *routGPU, *goutGPU, *boutGPU;

  hipMallocManaged(&routGPU, N*sizeof(int));
  hipMallocManaged(&goutGPU, N*sizeof(int));
  hipMallocManaged(&boutGPU, N*sizeof(int));

   // initialize x and y arrays on the host
   for (int i = 0; i < N; i++) {
    routGPU[i] = 0;
    goutGPU[i] = 0;
    boutGPU[i] = 0;

  }
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  renderPixel<<<numBlocks,blockSize>>>(N,routGPU,goutGPU,boutGPU);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  //copy memory to go
hipMemcpy(rout,routGPU,N*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(gout,goutGPU,N*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(bout,boutGPU,N*sizeof(int),hipMemcpyDeviceToHost);


   // Free memory
   hipFree(routGPU);
   hipFree(boutGPU);
   hipFree(goutGPU);
    
}